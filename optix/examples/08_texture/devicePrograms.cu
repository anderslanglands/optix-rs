#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>
#include <vec.h>

namespace osc {
#include "launch_params.h"
}

using namespace osc;

namespace osc {

/*! launch parameters in constant memory, filled in by optix upon
    optixLaunch (this gets filled in from the buffer we pass to
    optixLaunch) */
extern "C" __constant__ LaunchParams optixLaunchParams;

// for this simple example, we have a single ray type
enum { SURFACE_RAY_TYPE = 0, RAY_TYPE_COUNT };

static __forceinline__ __device__ void* unpackPointer(u32 i0, u32 i1) {
    const u64 uptr = static_cast<u64>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__ void packPointer(void* ptr, u32& i0,
                                                   u32& i1) {
    const u64 uptr = reinterpret_cast<u64>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template <typename T> static __forceinline__ __device__ T* getPRD() {
    const u32 u0 = optixGetPayload_0();
    const u32 u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

/*! helper function that creates a semi-random color from an ID */
inline __device__ V3f32 randomColor(int i) {
    int r = unsigned(i) * 13 * 17 + 0x234235;
    int g = unsigned(i) * 7 * 3 * 5 + 0x773477;
    int b = unsigned(i) * 11 * 19 + 0x223766;
    return V3f32((r & 255) / 255.f, (g & 255) / 255.f, (b & 255) / 255.f);
}

//------------------------------------------------------------------------------
// closest hit and anyhit programs for radiance-type rays.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but this
// simple example doesn't use any actual geometries yet, so we only
// create a single, dummy, set of them (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------
extern "C" __global__ void __closesthit__radiance() {
    const TriangleMeshSBTData& sbtData =
        *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // ------------------------------------------------------------------
    // gather some basic hit information
    // ------------------------------------------------------------------
    const int primID = optixGetPrimitiveIndex();
    const V3i32 index = sbtData.index[primID];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // ------------------------------------------------------------------
    // compute normal, using either shading normal (if avail), or
    // geometry normal (fallback)
    // ------------------------------------------------------------------
    V3f32 N(0.0f, 0.0f, 0.0f);
    if (!sbtData.normal.is_null()) {
        N = (1.f - u - v) * sbtData.normal[index.x] +
            u * sbtData.normal[index.y] + v * sbtData.normal[index.z];
    } else {
        const V3f32& A = sbtData.vertex[index.x];
        const V3f32& B = sbtData.vertex[index.y];
        const V3f32& C = sbtData.vertex[index.z];
        N = normalize(cross(B - A, C - A));
    }
    N = normalize(N);

    // ------------------------------------------------------------------
    // compute diffuse material color, including diffuse texture, if
    // available
    // ------------------------------------------------------------------
    V3f32 diffuseColor = sbtData.color;
    if (sbtData.has_texture && !sbtData.texcoord.is_null()) {
        const V2f32 tc = (1.f - u - v) * sbtData.texcoord[index.x] +
                         u * sbtData.texcoord[index.y] +
                         v * sbtData.texcoord[index.z];

        V4f32 fromTexture = tex2D<float4>(sbtData.texture, tc.x, 1.0f - tc.y);
        diffuseColor = diffuseColor * fromTexture.xyz();
    }

    // ------------------------------------------------------------------
    // perform some simple "NdotD" shading
    // ------------------------------------------------------------------
    const V3f32 rayDir = optixGetWorldRayDirection();
    const float cosDN = 0.2f + .8f * fabsf(dot(rayDir, N));

    V3f32& prd = *(V3f32*)getPRD<V3f32>();
    prd = cosDN * diffuseColor;
}

extern "C" __global__ void
__anyhit__radiance() { /*! for this simple example, this will remain empty */
}

//------------------------------------------------------------------------------
// miss program that gets called for any ray that did not have a
// valid intersection
//
// as with the anyhit/closest hit programs, in this example we only
// need to have _some_ dummy function to set up a valid SBT
// ------------------------------------------------------------------------------

extern "C" __global__ void __miss__radiance() {
    V3f32& prd = *(V3f32*)getPRD<V3f32>();
    // set to constant white as background color
    prd = V3f32(1.f, 1.0f, 1.0f);
}

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame() {
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;

    const auto& camera = optixLaunchParams.camera;

    // our per-ray data for this example. what we initialize it to
    // won't matter, since this value will be overwritten by either
    // the miss or hit program, anyway
    V3f32 pixelColorPRD = V3f32(0.f, 0.0f, 0.0f);

    // the values we store the PRD pointer in:
    u32 u0, u1;
    packPointer(&pixelColorPRD, u0, u1);

    // normalized screen plane position, in [0,1]^2
    const V2f32 screen =
        V2f32(f32(ix) + .5f, f32(iy) + .5f) /
        V2f32(optixLaunchParams.frame.size.x, optixLaunchParams.frame.size.y);

    // generate ray direction
    V3f32 rayDir =
        normalize(camera.direction + (screen.x - 0.5f) * camera.horizontal +
                  (screen.y - 0.5f) * camera.vertical);

    optixTrace(optixLaunchParams.traversable, (float3)camera.position,
               (float3)rayDir,
               0.f,   // tmin
               1e20f, // tmax
               0.0f,  // rayTime
               OptixVisibilityMask(255),
               OPTIX_RAY_FLAG_DISABLE_ANYHIT, // OPTIX_RAY_FLAG_NONE,
               SURFACE_RAY_TYPE,              // SBT offset
               RAY_TYPE_COUNT,                // SBT stride
               SURFACE_RAY_TYPE,              // missSBTIndex
               u0, u1);

    // and write to frame buffer ...
    const u32 fbIndex = ix + iy * optixLaunchParams.frame.size.x;
    optixLaunchParams.frame.color_buffer[fbIndex] =
        make_float4(pixelColorPRD.x, pixelColorPRD.y, pixelColorPRD.z, 1.0f);
}

} // namespace osc
