#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>
#include <vec.h>

namespace osc {
#include "launch_params.h"
}

using namespace osc;

namespace osc {

/*! launch parameters in constant memory, filled in by optix upon
    optixLaunch (this gets filled in from the buffer we pass to
    optixLaunch) */
extern "C" __constant__ LaunchParams optixLaunchParams;

// for this simple example, we have a single ray type
enum { SURFACE_RAY_TYPE = 0, RAY_TYPE_COUNT };

static __forceinline__ __device__ void* unpackPointer(u32 i0, u32 i1) {
    const u64 uptr = static_cast<u64>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__ void packPointer(void* ptr, u32& i0,
                                                   u32& i1) {
    const u64 uptr = reinterpret_cast<u64>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template <typename T> static __forceinline__ __device__ T* getPRD() {
    const u32 u0 = optixGetPayload_0();
    const u32 u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

/*! helper function that creates a semi-random color from an ID */
inline __device__ f32x3 randomColor(int i) {
    int r = unsigned(i) * 13 * 17 + 0x234235;
    int g = unsigned(i) * 7 * 3 * 5 + 0x773477;
    int b = unsigned(i) * 11 * 19 + 0x223766;
    return make_f32x3((r & 255) / 255.f, (g & 255) / 255.f, (b & 255) / 255.f);
}

//------------------------------------------------------------------------------
// closest hit and anyhit programs for radiance-type rays.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but this
// simple example doesn't use any actual geometries yet, so we only
// create a single, dummy, set of them (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------
extern "C" __global__ void __closesthit__radiance() {
    const TriangleMeshSBTData& sbtData =
        *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // ------------------------------------------------------------------
    // gather some basic hit information
    // ------------------------------------------------------------------
    const int primID = optixGetPrimitiveIndex();
    const i32x3 index = sbtData.index[primID];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // ------------------------------------------------------------------
    // compute normal, using either shading normal (if avail), or
    // geometry normal (fallback)
    // ------------------------------------------------------------------
    auto N = make_f32x3(0.0f, 0.0f, 0.0f);
    if (!sbtData.normal.is_null()) {
        N = (1.f - u - v) * sbtData.normal[index.x] +
            u * sbtData.normal[index.y] + v * sbtData.normal[index.z];
    } else {
        const f32x3& A = sbtData.vertex[index.x];
        const f32x3& B = sbtData.vertex[index.y];
        const f32x3& C = sbtData.vertex[index.z];
        N = normalize(cross(B - A, C - A));
    }
    N = normalize(N);

    // ------------------------------------------------------------------
    // compute diffuse material color, including diffuse texture, if
    // available
    // ------------------------------------------------------------------
    f32x3 diffuseColor = sbtData.color;
    if (sbtData.has_texture && !sbtData.texcoord.is_null()) {
        const f32x2 tc = (1.f - u - v) * sbtData.texcoord[index.x] +
                         u * sbtData.texcoord[index.y] +
                         v * sbtData.texcoord[index.z];

        f32x4 fromTexture = tex2D<float4>(sbtData.texture, tc.x, 1.0f - tc.y);
        diffuseColor = diffuseColor * make_f32x3(fromTexture);
    }

    // ------------------------------------------------------------------
    // perform some simple "NdotD" shading
    // ------------------------------------------------------------------
    const f32x3 rayDir = optixGetWorldRayDirection();
    const float cosDN = 0.2f + .8f * fabsf(dot(rayDir, N));

    f32x3& prd = *(f32x3*)getPRD<f32x3>();
    prd = cosDN * diffuseColor;
}

extern "C" __global__ void
__anyhit__radiance() { /*! for this simple example, this will remain empty */
}

//------------------------------------------------------------------------------
// miss program that gets called for any ray that did not have a
// valid intersection
//
// as with the anyhit/closest hit programs, in this example we only
// need to have _some_ dummy function to set up a valid SBT
// ------------------------------------------------------------------------------

extern "C" __global__ void __miss__radiance() {
    f32x3& prd = *(f32x3*)getPRD<f32x3>();
    // set to constant white as background color
    prd = make_f32x3(1.f, 1.0f, 1.0f);
}

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame() {
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;

    const auto& camera = optixLaunchParams.camera;

    // our per-ray data for this example. what we initialize it to
    // won't matter, since this value will be overwritten by either
    // the miss or hit program, anyway
    f32x3 pixelColorPRD = make_f32x3(0.f, 0.0f, 0.0f);

    // the values we store the PRD pointer in:
    u32 u0, u1;
    packPointer(&pixelColorPRD, u0, u1);

    // normalized screen plane position, in [0,1]^2
    const f32x2 screen = make_f32x2(f32(ix) + .5f, f32(iy) + .5f) /
                         make_f32x2(optixLaunchParams.frame.size.x,
                                    optixLaunchParams.frame.size.y);

    // generate ray direction
    f32x3 rayDir =
        normalize(camera.direction + (screen.x - 0.5f) * camera.horizontal +
                  (screen.y - 0.5f) * camera.vertical);

    optixTrace(optixLaunchParams.traversable, (float3)camera.position,
               (float3)rayDir,
               0.f,   // tmin
               1e20f, // tmax
               0.0f,  // rayTime
               OptixVisibilityMask(255),
               OPTIX_RAY_FLAG_DISABLE_ANYHIT, // OPTIX_RAY_FLAG_NONE,
               SURFACE_RAY_TYPE,              // SBT offset
               RAY_TYPE_COUNT,                // SBT stride
               SURFACE_RAY_TYPE,              // missSBTIndex
               u0, u1);

    // and write to frame buffer ...
    const u32 fbIndex = ix + iy * optixLaunchParams.frame.size.x;
    optixLaunchParams.frame.color_buffer[fbIndex] =
        make_float4(pixelColorPRD.x, pixelColorPRD.y, pixelColorPRD.z, 1.0f);
}

} // namespace osc
