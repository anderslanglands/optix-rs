#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <hip/hip_runtime.h>
#include <optix_device.h>

#include "lcg.h"
#include "vec.h"

enum { RADIANCE_RAY_TYPE = 0, SHADOW_RAY_TYPE, RAY_TYPE_COUNT };

namespace osc {
#include "launch_params.h"
}

using namespace osc;

#define NUM_LIGHT_SAMPLES 16
#define NUM_PIXEL_SAMPLES 4

namespace osc {

typedef LCG<16> Random;

/*! launch parameters in constant memory, filled in by optix upon
    optixLaunch (this gets filled in from the buffer we pass to
    optixLaunch) */
extern "C" __constant__ LaunchParams optixLaunchParams;

/*! per-ray data now captures random numebr generator, so programs
    can access RNG state */
struct PRD {
    Random random;
    f32x3 pixelColor;
};

static __forceinline__ DEVICE void* unpackPointer(u32 i0, u32 i1) {
    const u64 uptr = static_cast<u64>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ DEVICE void packPointer(void* ptr, u32& i0, u32& i1) {
    const u64 uptr = reinterpret_cast<u64>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template <typename T> static __forceinline__ DEVICE T* getPRD() {
    const u32 u0 = optixGetPayload_0();
    const u32 u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

//------------------------------------------------------------------------------
// closest hit and anyhit programs for radiance-type rays.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but this
// simple example doesn't use any actual geometries yet, so we only
// create a single, dummy, set of them (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------

extern "C" __global__ void __closesthit__shadow() {
    /* not going to be used ... */
}

extern "C" __global__ void __closesthit__radiance() {
    const TriangleMeshSBTData& sbtData =
        *(const TriangleMeshSBTData*)optixGetSbtDataPointer();
    PRD& prd = *getPRD<PRD>();

    // ------------------------------------------------------------------
    // gather some basic hit information
    // ------------------------------------------------------------------
    const i32 primID = optixGetPrimitiveIndex();
    const i32x3 index = sbtData.index[primID];
    const f32 u = optixGetTriangleBarycentrics().x;
    const f32 v = optixGetTriangleBarycentrics().y;

    // ------------------------------------------------------------------
    // compute normal, using either shading normal (if avail), or
    // geometry normal (fallback)
    // ------------------------------------------------------------------
    const f32x3& A = sbtData.vertex[index.x];
    const f32x3& B = sbtData.vertex[index.y];
    const f32x3& C = sbtData.vertex[index.z];
    f32x3 Ng = cross(B - A, C - A);
    f32x3 Ns =
        (sbtData.normal.is_null())
            ? Ng
            : ((1.f - u - v) * sbtData.normal[index.x] +
               u * sbtData.normal[index.y] + v * sbtData.normal[index.z]);

    // ------------------------------------------------------------------
    // face-forward and normalize normals
    // ------------------------------------------------------------------
    const f32x3 rayDir = optixGetWorldRayDirection();

    if (dot(rayDir, Ng) > 0.f)
        Ng = -Ng;
    Ng = normalize(Ng);

    if (dot(Ng, Ns) < 0.f)
        Ns = Ns - 2.f * dot(Ng, Ns) * Ng;
    Ns = normalize(Ns);

    // ------------------------------------------------------------------
    // compute diffuse material color, including diffuse texture, if
    // available
    // ------------------------------------------------------------------
    f32x3 diffuseColor = sbtData.color;
    if (sbtData.has_texture && !sbtData.texcoord.is_null()) {
        const f32x2 tc = (1.f - u - v) * sbtData.texcoord[index.x] +
                         u * sbtData.texcoord[index.y] +
                         v * sbtData.texcoord[index.z];

        f32x4 fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
        diffuseColor = diffuseColor * make_f32x3(fromTexture);
    }

    // start with some ambient term
    f32x3 pixelColor = (0.01f + 0.1f * fabsf(dot(Ns, rayDir))) * diffuseColor;

    // ------------------------------------------------------------------
    // compute shadow
    // ------------------------------------------------------------------
    const f32x3 surfPos = (1.f - u - v) * sbtData.vertex[index.x] +
                          u * sbtData.vertex[index.y] +
                          v * sbtData.vertex[index.z];

    const i32 numLightSamples = NUM_LIGHT_SAMPLES;
    for (i32 lightSampleID = 0; lightSampleID < numLightSamples;
         lightSampleID++) {
        // produce random light sample
        const f32x3 lightPos = optixLaunchParams.light.origin +
                               prd.random() * optixLaunchParams.light.du +
                               prd.random() * optixLaunchParams.light.dv;
        f32x3 lightDir = lightPos - surfPos;
        f32 lightDist = length(lightDir);
        lightDir = normalize(lightDir);

        // trace shadow ray:
        const f32 NdotL = dot(lightDir, Ns);
        if (NdotL >= 0.f) {
            f32x3 lightVisibility=make_float3(1.f,1.f,1.f);
            // the values we store the PRD poi32er in:
            u32 u0, u1;
            packPointer(&lightVisibility, u0, u1);
            optixTrace(optixLaunchParams.traversable, surfPos + 1e-3f * Ng,
                       lightDir,
                       1e-3f,                     // tmin
                       lightDist * (1.f - 1e-3f), // tmax
                       0.0f,                      // rayTime
                       OptixVisibilityMask(255),
                       // anyhit ON for shadow rays:
                       OPTIX_RAY_FLAG_NONE,
                       SHADOW_RAY_TYPE, // SBT offset
                       RAY_TYPE_COUNT,  // SBT stride
                       SHADOW_RAY_TYPE, // missSBTIndex
                       u0, u1);
            pixelColor =
                pixelColor +
                lightVisibility * optixLaunchParams.light.power * diffuseColor *
                    (NdotL / (lightDist * lightDist * numLightSamples));
        }
    }

    prd.pixelColor = pixelColor;
}

extern "C" __global__ void
__anyhit__radiance() { /*! for this simple example, this will remain empty */
}

extern "C" __global__ void __anyhit__shadow() {
    // in this simple example, we terminate on ANY hit
    f32x3& prd = *getPRD<f32x3>();
    prd = make_float3(0.f,0.f,0.f);
    optixTerminateRay();
}

//------------------------------------------------------------------------------
// miss program that gets called for any ray that did not have a
// valid i32ersection
//
// as with the anyhit/closest hit programs, in this example we only
// need to have _some_ dummy function to set up a valid SBT
// ------------------------------------------------------------------------------

extern "C" __global__ void __miss__radiance() {
    PRD& prd = *getPRD<PRD>();
    // set to constant white as background color
    prd.pixelColor = make_float3(1.f,1.f,1.f);
}

extern "C" __global__ void __miss__shadow() {
    // misses shouldn't mess with shadow opacity - do nothing
}

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame() {
    // compute a test pattern based on pixel ID
    const i32 ix = optixGetLaunchIndex().x;
    const i32 iy = optixGetLaunchIndex().y;
    const i32 accum_id = optixLaunchParams.frame.accum_id;
    const auto& camera = optixLaunchParams.camera;

    PRD prd;
    prd.random.init(ix + accum_id * optixLaunchParams.frame.size.x,
                    iy + accum_id * optixLaunchParams.frame.size.y);
    prd.pixelColor = make_float3(0.f,0.f,0.f);

    // the values we store the PRD poi32er in:
    u32 u0, u1;
    packPointer(&prd, u0, u1);

    i32 numPixelSamples = NUM_PIXEL_SAMPLES;

    f32x3 pixelColor=make_float3(0.f,0.f,0.f);
    for (i32 sampleID = 0; sampleID < numPixelSamples; sampleID++) {
        // normalized screen plane position, in [0,1]^2
        const f32x2 screen(make_float2(ix + prd.random(), iy + prd.random()) /
                           make_float2(optixLaunchParams.frame.size.x,
                                 optixLaunchParams.frame.size.y));

        // generate ray direction
        f32x3 rayDir =
            normalize(camera.direction + (screen.x - 0.5f) * camera.horizontal +
                      (screen.y - 0.5f) * camera.vertical);

        optixTrace(optixLaunchParams.traversable, camera.position, rayDir,
                   0.f,   // tmin
                   1e20f, // tmax
                   0.0f,  // rayTime
                   OptixVisibilityMask(255),
                   OPTIX_RAY_FLAG_DISABLE_ANYHIT, // OPTIX_RAY_FLAG_NONE,
                   RADIANCE_RAY_TYPE,             // SBT offset
                   RAY_TYPE_COUNT,                // SBT stride
                   RADIANCE_RAY_TYPE,             // missSBTIndex
                   u0, u1);
        pixelColor = pixelColor + prd.pixelColor;
    }

    const u32 fbIndex = ix + iy * optixLaunchParams.frame.size.x;
    optixLaunchParams.frame.color_buffer[fbIndex] = make_float4(
        pixelColor.x / numPixelSamples, pixelColor.y / numPixelSamples,
        pixelColor.z / numPixelSamples, 1.0f);
}

} // namespace osc
