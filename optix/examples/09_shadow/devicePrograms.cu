#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>
#include <vec.h>

#include "LaunchParams.h"

using namespace osc;

namespace osc {

/*! launch parameters in constant memory, filled in by optix upon
    optixLaunch (this gets filled in from the buffer we pass to
    optixLaunch) */
extern "C" __constant__ LaunchParams optixLaunchParams;

static __forceinline__ __device__ void* unpackPointer(u32 i0, u32 i1) {
    const u64 uptr = static_cast<u64>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__ void packPointer(void* ptr, u32& i0,
                                                   u32& i1) {
    const u64 uptr = reinterpret_cast<u64>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template <typename T> static __forceinline__ __device__ T* getPRD() {
    const u32 u0 = optixGetPayload_0();
    const u32 u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

//------------------------------------------------------------------------------
// closest hit and anyhit programs for radiance-type rays.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but this
// simple example doesn't use any actual geometries yet, so we only
// create a single, dummy, set of them (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------
extern "C" __global__ void __closesthit__shadow() {
    /* not going to be used ... */
}

extern "C" __global__ void __closesthit__radiance() {
    const TriangleMeshSBTData& sbtData =
        *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // ------------------------------------------------------------------
    // gather some basic hit information
    // ------------------------------------------------------------------
    const int primID = optixGetPrimitiveIndex();
    const V3i32 index = sbtData.index[primID];
    const f32 u = optixGetTriangleBarycentrics().x;
    const f32 v = optixGetTriangleBarycentrics().y;

    // ------------------------------------------------------------------
    // compute normal, using either shading normal (if avail), or
    // geometry normal (fallback)
    // ------------------------------------------------------------------
    const V3f32& A = sbtData.vertex[index.x];
    const V3f32& B = sbtData.vertex[index.y];
    const V3f32& C = sbtData.vertex[index.z];
    V3f32 Ng = cross(B - A, C - A);
    V3f32 Ns = (sbtData.normal)
                   ? ((1.f - u - v) * sbtData.normal[index.x] +
                      u * sbtData.normal[index.y] + v * sbtData.normal[index.z])
                   : Ng;

    // ------------------------------------------------------------------
    // face-forward and normalize normals
    // ------------------------------------------------------------------
    const V3f32 rayDir = optixGetWorldRayDirection();

    if (dot(rayDir, Ng) > 0.f)
        Ng = -Ng;
    Ng = normalize(Ng);

    if (dot(Ng, Ns) < 0.f)
        Ns = Ns - 2.f * dot(Ng, Ns) * Ng;
    Ns = normalize(Ns);

    // ------------------------------------------------------------------
    // compute diffuse material color, including diffuse texture, if
    // available
    // ------------------------------------------------------------------
    V3f32 diffuseColor = sbtData.color;
    if (sbtData.hasTexture && sbtData.texcoord) {
        const V2f32 tc = (1.f - u - v) * sbtData.texcoord[index.x] +
                         u * sbtData.texcoord[index.y] +
                         v * sbtData.texcoord[index.z];

        V4f32 fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
        diffuseColor = diffuseColor * fromTexture.xyz();
    }

    // ------------------------------------------------------------------
    // compute shadow
    // ------------------------------------------------------------------
    const V3f32 surfPos = (1.f - u - v) * sbtData.vertex[index.x] +
                          u * sbtData.vertex[index.y] +
                          v * sbtData.vertex[index.z];
    const V3f32 lightPos(-907.108f, 2205.875f, -400.0267f);
    const V3f32 lightDir = lightPos - surfPos;

    // trace shadow ray:
    V3f32 lightVisibility(1.f);
    // the values we store the PRD pointer in:
    u32 u0, u1;
    packPointer(&lightVisibility, u0, u1);
    optixTrace(optixLaunchParams.traversable, surfPos + 1e-3f * Ng, lightDir,
               1e-3f,       // tmin
               1.f - 1e-3f, // tmax
               0.0f,        // rayTime
               OptixVisibilityMask(255),
               // anyhit ON for shadow rays:
               OPTIX_RAY_FLAG_NONE,
               SHADOW_RAY_TYPE, // SBT offset
               RAY_TYPE_COUNT,  // SBT stride
               SHADOW_RAY_TYPE, // missSBTIndex
               u0, u1);

    // ------------------------------------------------------------------
    // final shading: a bit of ambient, a bit of directional ambient,
    // and directional component based on shadowing
    // ------------------------------------------------------------------
    const float cosDN = 0.1f + .8f * fabsf(dot(rayDir, Ns));

    V3f32& prd = *(V3f32*)getPRD<V3f32>();
    prd = (.1f + (.2f + .8f * lightVisibility) * cosDN) * diffuseColor;
}

extern "C" __global__ void
__anyhit__radiance() { /*! for this simple example, this will remain empty */
}

extern "C" __global__ void __anyhit__shadow() {
    // in this simple example, we terminate on ANY hit
    V3f32& prd = *(V3f32*)getPRD<V3f32>();
    prd = V3f32(0.f);
    optixTerminateRay();
}

//------------------------------------------------------------------------------
// miss program that gets called for any ray that did not have a
// valid intersection
//
// as with the anyhit/closest hit programs, in this example we only
// need to have _some_ dummy function to set up a valid SBT
// ------------------------------------------------------------------------------

extern "C" __global__ void __miss__radiance() {
    V3f32& prd = *(V3f32*)getPRD<V3f32>();
    // set to constant white as background color
    prd = V3f32(1.f);
}

extern "C" __global__ void __miss__shadow() {
    // misses shouldn't mess with shadow opacity - do nothing
}

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__renderFrame() {
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;

    const auto& camera = optixLaunchParams.camera;

    // our per-ray data for this example. what we initialize it to
    // won't matter, since this value will be overwritten by either
    // the miss or hit program, anyway
    V3f32 pixelColorPRD = V3f32(0.f, 0.0f, 0.0f);

    // the values we store the PRD pointer in:
    u32 u0, u1;
    packPointer(&pixelColorPRD, u0, u1);

    // normalized screen plane position, in [0,1]^2
    const V2f32 screen =
        V2f32(f32(ix) + .5f, f32(iy) + .5f) /
        V2f32(optixLaunchParams.frame.size.x, optixLaunchParams.frame.size.y);

    // generate ray direction
    V3f32 rayDir =
        normalize(camera.direction + (screen.x - 0.5f) * camera.horizontal +
                  (screen.y - 0.5f) * camera.vertical);

    optixTrace(optixLaunchParams.traversable, (float3)camera.position,
               (float3)rayDir,
               0.f,   // tmin
               1e20f, // tmax
               0.0f,  // rayTime
               OptixVisibilityMask(255),
               OPTIX_RAY_FLAG_DISABLE_ANYHIT, // OPTIX_RAY_FLAG_NONE,
               RADIANCE_RAY_TYPE,             // SBT offset
               RAY_TYPE_COUNT,                // SBT stride
               RADIANCE_RAY_TYPE,             // missSBTIndex
               u0, u1);

    // and write to frame buffer ...
    const u32 fbIndex = ix + iy * optixLaunchParams.frame.size.x;
    optixLaunchParams.frame.colorBuffer[fbIndex] =
        make_float4(pixelColorPRD.x, pixelColorPRD.y, pixelColorPRD.z, 1.0f);
}

} // namespace osc
